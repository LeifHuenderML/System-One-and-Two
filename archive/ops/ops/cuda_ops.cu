#include "hip/hip_runtime.h"
#include "andrea/ops/cuda_ops.hpp"


namespace andrea{

__global__ void add_kernel(float* a, float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

Tensor* cuda_add(const Tensor* a, const Tensor* b) {
    if (a->device != "cuda" || b->device != "cuda") {
        throw std::runtime_error("Both tensors must be on CUDA device for cuda_add operation.");
    }
    if (a->size != b->size) {
        throw std::runtime_error("Tensor sizes must match for addition.");
    }

    Tensor* result = create_tensor(nullptr, std::vector<int>(a->shape, a->shape + a->ndim), "cuda");

    int block_size = 256;
    int num_blocks = (a->size + block_size - 1) / block_size;

    add_kernel<<<num_blocks, block_size>>>(a->data, b->data, result->data, a->size);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return result;
}

} // namespace andrea
