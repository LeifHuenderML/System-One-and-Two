// hello_cuda.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    int a = 0;
    while(true){
        a = 100/3;
    }

}

int main() {
    // Launch kernel
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize(); // Wait for the GPU to finish
    return 0;
}
